#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define SIZE(A) A*sizeof(int)
#define FSIZE(A) A*sizeof(float)
#define LENGTH 8 

// Test the new SST instruction's functionality
__global__ void SSTTest(float* V, int* addr, int N) {
	int i = threadIdx.x;
	if (i < N) {
		int return_val = 0;
		float element = V[i];
		asm("/*");
		asm("CPTX_BEGIN");
		asm(".sstarr .align 4 .b8 _Z9MatrixMulPfS_Pii__sst_var[32];"); // initialize sst_array
		asm("sst.sstarr.f32 %0, [%1], %2, %3;" : "=r"(return_val) : "l"(&V[0]), "r"(i), "f"(element)); // perform SST instruction
		asm("CPTX_END");
		asm("*/");
		if (i == N-1) *addr = return_val;
	}
}

int main(int argc, char** argv) {
	float *h_vector = (float*)calloc(2*LENGTH, FSIZE(1)); // twice as long to store values as well as indices
	int *h_addr = (int*)malloc(SIZE(1));

	for (int i = 0; i < LENGTH; i++) {
		if (i % 2 == 1) h_vector[i] = 0.0;
		else h_vector[i] = i + 1.0;
		h_vector[i+LENGTH] = 0.0; // second half of the array is initially all 0
	}

	float *d_vector;
	int *d_addr;
	hipMalloc(&d_vector, FSIZE(2*LENGTH));
	hipMalloc(&d_addr, SIZE(1));
	hipMemcpy(d_vector, h_vector, FSIZE(2*LENGTH), hipMemcpyHostToDevice);
	SSTTest<<<1, LENGTH>>>(d_vector, d_addr, LENGTH);
	hipMemcpy(h_vector, d_vector, FSIZE(2*LENGTH), hipMemcpyDeviceToHost);
	hipMemcpy(h_addr, d_addr, SIZE(1), hipMemcpyDeviceToHost);
	
	// output results
	printf("\nValues: \tIndices:\n");
	for (int i = 0; i < LENGTH; i++) {
		printf("%f\t%d\n", h_vector[i], (int)h_vector[i+LENGTH]);
	}
	printf("\nSST return value: %d\n", h_addr[0]);
	printf("Array start: %p\nArray end: %p\n", &h_vector[0], &h_vector[h_addr[0]]);

	hipFree(d_vector);
	hipFree(d_addr);
	free(h_vector);
	free(h_addr);

	return 0;
}
