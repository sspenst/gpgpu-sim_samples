#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define THREADS 8
#define IP 4
#define OP 4
#define INBUFFERS 4
#define OUTBUFFERS ((((32/IP)*INBUFFERS) / (32/OP)) + ((((32/IP)*INBUFFERS) % (32/OP)) != 0))

__global__ void BSMAD(int *buffer, int *synapse, int *output) {
	volatile int tid = threadIdx.x;
	volatile int result = 0;
	asm("/*");
	asm("CPTX_BEGIN");
	asm("bsmad.s32 %0, %1, %2, %3, %4, %5, %6, %7, %8;" : "=r"(result) : "r"(IP), "r"(OP), "r"(buffer[tid*INBUFFERS]), "r"(buffer[tid*INBUFFERS+1]), "r"(buffer[tid*INBUFFERS+2]), "r"(buffer[tid*INBUFFERS+3]), "r"(synapse[tid]), "r"(output[tid%OUTBUFFERS]));
	asm("CPTX_END");
	asm("*/");
	if (tid < OUTBUFFERS) output[tid] = result;
}

int main(int argc, char** argv) {
	int *h_buffer = (int*)malloc(INBUFFERS*THREADS*sizeof(int));
	int *h_synapse = (int*)malloc(THREADS*sizeof(int));
	int *h_output = (int*)calloc(OUTBUFFERS,sizeof(int));

	for (int i = 0; i < INBUFFERS*THREADS; i++) { h_buffer[i] = 0xFDB97531; }
	for (int i = 0; i < THREADS; i++) { h_synapse[i] = 1; }
	for (int i = 0; i < OUTBUFFERS; i++) { h_output[i] = 0x87654321; }

	int *d_buffer, *d_synapse, *d_output;
	hipMalloc(&d_buffer, INBUFFERS*THREADS*sizeof(int));
	hipMalloc(&d_synapse, THREADS*sizeof(int));
	hipMalloc(&d_output, OUTBUFFERS*sizeof(int));
	hipMemcpy(d_buffer, h_buffer, INBUFFERS*THREADS*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_synapse, h_synapse, THREADS*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_output, h_output, OUTBUFFERS*sizeof(int), hipMemcpyHostToDevice);
	
	BSMAD<<<1, THREADS>>>(d_buffer, d_synapse, d_output);
	
	hipMemcpy(h_output, d_output, OUTBUFFERS*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < OUTBUFFERS; i++) { printf("%d: %08x\n", i, h_output[i]); }

	hipFree(d_buffer); hipFree(d_synapse); hipFree(d_output);
	free(h_buffer); free(h_synapse); free(h_output);

	return 0;
}
